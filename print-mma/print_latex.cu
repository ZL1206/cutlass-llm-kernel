#include <iostream>
#include <hip/hip_runtime.h>
#include <cute/tensor.hpp>
#include <iostream>
#include <iomanip>
#include <utility>
#include <type_traits>
#include <vector>
#include <numeric>
#include <cutlass/cutlass.h>
#include <cutlass/numeric_types.h>
#include <cutlass/trace.h>



int main(void) {
    using namespace cute;
    using T = cute::half_t;
    
    

    using TiledMma = TiledMMA<
        MMA_Atom<SM80_16x8x16_F32F16F16F32_TN>,
        Layout<Shape<Int<4>,_1,_1>>,  // 4x1x1 or 8x1x1 thread group
        Tile<Int<16 * 4>, _16, _16>>;
    
    using A = Layout< Shape<Shape<Int<2>, Int<2>, Int<2>>, Int<2>, Shape<Shape<Int<2>, Int<2>>, Int<2>>>,
                     Stride<Stride<Int<1>, Int<0>>, Int<2048>> >;

    
    //print_latex(A{});
    print_layout(A{});
    
    
    return 0;
}

