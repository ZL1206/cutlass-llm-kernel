#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <cute/tensor.hpp>
#include <iostream>
#include <iomanip>
#include <utility>
#include <type_traits>
#include <vector>
#include <random>
#include <numeric>
#include <cutlass/cutlass.h>
#include <cutlass/numeric_types.h>
#include <cute/tensor.hpp>
#include <cutlass/trace.h>


using namespace cute;

template <typename Kernel_traits>
__global__ void g2sCopy(void* q, void* k) {
    using T = typename Kernel_traits::T;
    constexpr int M = Kernel_traits::kTileM;
    constexpr int N = Kernel_traits::kTileN;
    constexpr int K = Kernel_traits::kTileK;

    using GmemTiledCopyQKV = typename Kernel_traits::GmemTiledCopyQKV;
    using SmemLayoutQ = typename Kernel_traits::SmemLayoutQ;
    using SmemLayoutKV = typename Kernel_traits::SmemLayoutKV;

    extern __shared__ char smem[];

    Tensor gQ = make_tensor(make_gmem_ptr(reinterpret_cast<T*>(q)),
                    make_shape(Int<M>{}, Int<K>{}),
                    make_stride(Int<K>{}, Int<1>{}));
    Tensor sQ = make_tensor(make_smem_ptr(reinterpret_cast<T*>(smem)),
                            SmemLayoutQ{});

    Tensor gK = make_tensor(make_gmem_ptr(reinterpret_cast<T*>(k)),
                    make_shape(Int<N>{}, Int<K>{}),
                    make_stride(Int<K>{}, Int<1>{}));
    
    Tensor sK = make_tensor(sQ.data() + size(sQ),
                            SmemLayoutKV{});
    
    const int idx = threadIdx.x;

    GmemTiledCopyQKV gmem_tiled_copy_QKV;
    auto gmem_thr_copy_QKV = gmem_tiled_copy_QKV.get_slice(idx);
    Tensor tQgQ = gmem_thr_copy_QKV.partition_S(gQ);
    Tensor tQsQ = gmem_thr_copy_QKV.partition_D(sQ);
    Tensor tKgK = gmem_thr_copy_QKV.partition_S(gK);  
    Tensor tKsK = gmem_thr_copy_QKV.partition_D(sK);

    if (thread0()) {
        print("tensor tQgQ: \n"); print_tensor(tQgQ); print("\n");
        print("tensor tKgK: \n"); print_tensor(tKgK); print("\n");
    }

    __syncthreads();

    if (thread0()){
        print("gmem_thr_copy_QKV: \n"); print(gmem_thr_copy_QKV); print("\n");
        print("sQ: \n"); print(sQ); print("\n");
        print("tQgQ: \n"); print(tQgQ); print("\n");
        print("tQsQ: \n"); print(tQsQ); print("\n");
        print("sK: \n"); print(sK); print("\n");
        print("tKgK: \n"); print(tKgK); print("\n");
        print("tKsK: \n"); print(tKsK); print("\n");
    }
    
    // global to shared memory
    for (int m = 0; m < size<1>(tQgQ); m++) {
        for (int k = 0; k < size<2>(tQgQ); k++) {
            copy(gmem_tiled_copy_QKV, tQgQ(_, m, k), tQsQ(_, m, k));
        }
    }

    for (int m = 0; m < size<1>(tKgK); m++) {
        for (int k = 0; k < size<2>(tKgK); k++) {
            copy(gmem_tiled_copy_QKV, tKgK(_, m, k), tKsK(_, m, k));
        }
    }

    cp_async_fence();

    cp_async_wait<0>();

    __syncthreads();

    if (thread0()) {

        printf("Q -----------------------------------------------------------------------------------------------------\n");
        print("tensor sQ: \n"); print_tensor(sQ); print("\n");
        printf("real sQ: \n");
        
        T* smem_ptr = reinterpret_cast<T*>(smem);
        for (int i = 0; i < size(sQ); i++) {
            float tmp = static_cast<float>(smem_ptr[i]);
            printf("%10.6f", tmp); printf(" ");
            if ((i + 1) % 8 == 0 && (i + 1) % 64 != 0) {
                printf("|"); printf(" ");
            }
            if ((i+1) % 64 == 0) {
                printf("\n");
            }
        }


        printf("K -----------------------------------------------------------------------------------------------------\n");
        print("tensor sK: \n"); print_tensor(sK); print("\n");
        printf("real sK: \n");
        
        smem_ptr = smem_ptr + size(sQ);
        for (int i = 0; i < size(sK); i++) {
            float tmp = static_cast<float>(smem_ptr[i]);
            printf("%10.6f", tmp); printf(" ");
            if ((i + 1) % 8 == 0 && (i + 1) % 64 != 0) {
                printf("|"); printf(" ");
            }
            if ((i+1) % 64 == 0) {
                printf("\n");
            }
        }
        
    }
}



template <typename T_, int kTileM_ = 128, int kTileN_ = 32, int kTileK_ = 128>
struct Kernel_traits {

  using T = T_;

  // tile configuration
  static constexpr int kTileM = kTileM_;
  static constexpr int kTileN = kTileN_;
  static constexpr int kTileK = kTileK_;
  
  static constexpr int kShmLoadSwizzleM = 3;
  static constexpr int kShmLoadSwizzleS = 3;
  static constexpr int kShmLoadSwizzleB = 3; 

  // global to shared memory
  using GmemLayoutAtom = Layout<Shape <Int<16>, Int<8>>,
                                  Stride<Int<8>, _1>>;
    
  using GmemTiledCopyQKV = decltype(make_tiled_copy(Copy_Atom<SM80_CP_ASYNC_CACHEGLOBAL<cute::uint128_t>, T>{},
                        GmemLayoutAtom{},
                        Layout<Shape<_1, _8>>{}));
  // write o
  using GmemTiledCopyO = decltype(
        make_tiled_copy(Copy_Atom<AutoVectorizingCopyWithAssumedAlignment<128>, T>{},
                        GmemLayoutAtom{},
                        Layout<Shape<_1, _8>>{}));  // Val layout, 8 vals per store

  // shared memory layout
  /*
  using SmemLayoutAtomQ = decltype(
        composition(Swizzle<3, 3, 3>{},
                    // This has to be kBlockKSmem, using kHeadDim gives wrong results for d=128
                    Layout<Shape<_8, Int<64>>,
                           Stride<Int<64>, _1>>{}));
  */
  
  using SmemLayoutAtomQ = Layout<Shape<_8, Int<64>>,
                           Stride<Int<64>, _1>>;
  
  using SmemLayoutQ = decltype(tile_to_shape(
        SmemLayoutAtomQ{},
        Shape<Int<kTileM>, Int<kTileK>>{}));

  using SmemLayoutKV = decltype(tile_to_shape(
        SmemLayoutAtomQ{},
        Shape<Int<kTileN>, Int<kTileK>>{}));

  using SmemLayoutVtransposed = decltype(
        composition(SmemLayoutKV{}, make_layout(Shape<Int<kTileK>, Int<kTileN>>{}, GenRowMajor{})));
  using SmemLayoutVtransposedNoSwizzle = decltype(get_nonswizzle_portion(SmemLayoutVtransposed{}));

  using SmemLayoutAtomO = decltype(
        composition(Swizzle<3, 3, 3>{},
                    Layout<Shape<Int<8>, Int<64>>,
                           Stride<Int<64>, _1>>{}));
  using SmemLayoutO = decltype(tile_to_shape(
        SmemLayoutAtomO{},
        Shape<Int<kTileM>, Int<kTileK>>{}));

  // shared memory to register copy
  using SmemCopyAtom = Copy_Atom<SM75_U32x4_LDSM_N, T>;
  using SmemCopyAtomTransposed = Copy_Atom<SM75_U16x8_LDSM_T, T>; 

  
  using SmemCopyAtomO = Copy_Atom<DefaultCopy, T>;
  
  // tiled mma
  using TiledMma = TiledMMA<
        MMA_Atom<SM80_16x8x16_F32F16F16F32_TN>,
        Layout<Shape<Int<4>,_1,_1>>,  // 4x1x1 or 8x1x1 thread group
        Tile<Int<16 * 4>, _16, _16>>;
  
  static constexpr int kSmemQSize = size(SmemLayoutQ{}) * sizeof(T);
  static constexpr int kSmemKVSize = size(SmemLayoutKV{}) * 2 * sizeof(T);
  static constexpr int kSmemSize = kSmemQSize + kSmemKVSize;
};


int main(void) {
    using namespace cute;
    using T = cute::half_t;
    std::mt19937 gen(20250102);
    std::uniform_real_distribution<float> dis(static_cast<float>(-100), static_cast<float>(100));
    constexpr int M = 128;
    constexpr int N = 64;
    constexpr int K = 128;

    // q
    T* h_q = (T*)malloc(M * K * sizeof(T));
    for (int i = 0; i < M * K; i++) {
        float data = dis(gen);
        h_q[i] = T(data);
        printf("%10.6f", data); printf(" ");
        if ((i + 1) % 8 == 0 && (i+1) % 128 != 0) {
            printf("|"); printf(" ");
        }
        if ((i+1) % 128 == 0) {
            printf("\n");
        }
    }
    T* d_q = nullptr;
    hipMalloc(&d_q, M * K * sizeof(T));
    hipMemcpy(d_q, h_q, sizeof(T) * M * K, hipMemcpyHostToDevice);
    printf("-------------------------------------------------------------------\n");

    // k
    std::mt19937 genk(20250102 + 1);
    T* h_k = (T*)malloc(N * K * sizeof(T));
    for (int i = 0; i < N * K; i++) {
        float data = dis(genk);
        h_k[i] = T(data);
        printf("%10.6f", data); printf(" ");
        if ((i + 1) % 8 == 0 && (i+1) % 128 != 0) {
            printf("|"); printf(" ");
        }
        if ((i+1) % 128 == 0) {
            printf("\n");
        }
    }
    T* d_k = nullptr;
    hipMalloc(&d_k, N * K * sizeof(T));
    hipMemcpy(d_k, h_k, sizeof(T) * N * K, hipMemcpyHostToDevice);

    Kernel_traits<T, M, N, K> config;

    using SmemLayoutAtomQ = typename decltype(config)::SmemLayoutAtomQ;
    print("SmemLayoutAtomQ \n");
    print_layout(SmemLayoutAtomQ{});

    using SmemLayoutQ = typename decltype(config)::SmemLayoutQ;
    print("SmemLayoutQ \n");
    print_layout(SmemLayoutQ{});

    using SmemLayoutKV = typename decltype(config)::SmemLayoutKV;
    print("SmemLayoutKV \n");
    print_layout(SmemLayoutKV{});


    auto kernel = &g2sCopy<decltype(config)>;
    const int smem_size = config.kSmemSize;
    printf("smem_size is %d\n", smem_size);
    if (smem_size >= 48 * 1024) {
        hipFuncSetAttribute(reinterpret_cast<const void*>(kernel), hipFuncAttributeMaxDynamicSharedMemorySize, smem_size);
    }
    kernel<<<1, 128, smem_size>>>(d_q, d_k);
    
    hipDeviceSynchronize();
    auto err = hipGetLastError();
    printf("Copy done, Error Code: %d, State: %s\n", err, hipGetErrorString(err));

    if (d_q) {
        hipFree(d_q);
    }
    if (d_k) {
        hipFree(d_k);
    }
    return 0;
}