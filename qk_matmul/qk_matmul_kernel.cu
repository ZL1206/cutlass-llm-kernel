#include "hip/hip_runtime.h"
#include <iostream>
#include <torch/extension.h>
#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAGuard.h>
#include "static_switch.h"
#include "utils.h"
#include "softmax.h"
#include <hip/hip_runtime.h>
#include <cute/tensor.hpp>
#include <vector>
#include <random>
#include <numeric>
#include <cutlass/cutlass.h>
#include <cutlass/numeric_types.h>
#include <cutlass/numeric_conversion.h>
#include <cute/tensor.hpp>
#include <cutlass/trace.h>


using namespace cute;

template <typename T_, int kTileM_ = 128, int kTileN_ = 32, int kTileK_ = 128, int kNWarps_ = 4>
struct Kernel_traits {

  using T = T_;

  // tile configuration
  static constexpr int kTileM = kTileM_;
  static constexpr int kTileN = kTileN_;
  static constexpr int kTileK = kTileK_;

  static constexpr int kNWarps = kNWarps_;
  static constexpr int kNThreads = kNWarps * 32;
  
  static constexpr int kShmLoadSwizzleM = 3;
  static constexpr int kShmLoadSwizzleS = 3;
  static constexpr int kShmLoadSwizzleB = 3; 

  // global to shared memory
  using GmemLayoutAtom = Layout<Shape <Int<16>, Int<8>>,
                                  Stride<Int<8>, _1>>;
    
  using GmemTiledCopyQKV = decltype(make_tiled_copy(Copy_Atom<SM80_CP_ASYNC_CACHEGLOBAL<cute::uint128_t>, T>{},
                        GmemLayoutAtom{},
                        Layout<Shape<_1, _8>>{}));
  // write o
  using GmemTiledCopyO = decltype(
        make_tiled_copy(Copy_Atom<AutoVectorizingCopyWithAssumedAlignment<128>, T>{},
                        GmemLayoutAtom{},
                        Layout<Shape<_1, _8>>{}));  // Val layout, 8 vals per store

  // shared memory layout
  /*
  using SmemLayoutAtomQ = decltype(
        composition(Swizzle<3, 3, 3>{},
                    // This has to be kBlockKSmem, using kHeadDim gives wrong results for d=128
                    Layout<Shape<_8, Int<64>>,
                           Stride<Int<64>, _1>>{}));
  */
  using SmemLayoutAtomQ = Layout<Shape<_8, Int<64>>,
                           Stride<Int<64>, _1>>;
  
  
  using SmemLayoutQ = decltype(tile_to_shape(
        SmemLayoutAtomQ{},
        Shape<Int<kTileM>, Int<kTileK>>{}));

  using SmemLayoutKV = decltype(tile_to_shape(
        SmemLayoutAtomQ{},
        Shape<Int<kTileN>, Int<kTileK>>{}));

  using SmemLayoutVtransposed = decltype(
        composition(SmemLayoutKV{}, make_layout(Shape<Int<kTileK>, Int<kTileN>>{}, GenRowMajor{})));
  using SmemLayoutVtransposedNoSwizzle = decltype(get_nonswizzle_portion(SmemLayoutVtransposed{}));

  /*
  using SmemLayoutAtomO = decltype(
        composition(Swizzle<3, 3, 3>{},
                    Layout<Shape<Int<8>, Int<64>>,
                           Stride<Int<64>, _1>>{}));
  */
  using SmemLayoutAtomO = Layout<Shape<Int<8>, Int<64>>,
                           Stride<Int<64>, _1>>;
  using SmemLayoutO = decltype(tile_to_shape(
        SmemLayoutAtomO{},
        Shape<Int<kTileM>, Int<kTileN>>{}));

  // shared memory to register copy
  using SmemCopyAtom = Copy_Atom<SM75_U32x4_LDSM_N, T>;
  using SmemCopyAtomTransposed = Copy_Atom<SM75_U16x8_LDSM_T, T>; 

  
  using SmemCopyAtomO = Copy_Atom<AutoVectorizingCopyWithAssumedAlignment<128>, T>;
  
  // tiled mma
  using TiledMma = TiledMMA<
        MMA_Atom<SM80_16x8x16_F32F16F16F32_TN>,
        Layout<Shape<Int<kNWarps>,_1,_1>>,  // 4x1x1 or 8x1x1 thread group
        Tile<Int<16 * kNWarps>, _16, _16>>;
  
  static constexpr int kSmemQSize = size(SmemLayoutQ{}) * sizeof(T);
  static constexpr int kSmemKVSize = size(SmemLayoutKV{}) * 2 * sizeof(T);
  static constexpr int kSmemSize = kSmemQSize + kSmemKVSize;
};

struct fwd_params {
    void *__restrict__ q_ptr;
    void *__restrict__ k_ptr;
    void * __restrict__ o_ptr;
    float scale_softmax;
    float scale_softmax_log2;
};

template <typename Kernel_traits>
__global__ void qk_matmul_kernel(fwd_params params) {
    using T = typename Kernel_traits::T;
    constexpr int M = Kernel_traits::kTileM;
    constexpr int N = Kernel_traits::kTileN;
    constexpr int K = Kernel_traits::kTileK;

    using GmemTiledCopyQKV = typename Kernel_traits::GmemTiledCopyQKV;
    using SmemLayoutQ = typename Kernel_traits::SmemLayoutQ;
    using SmemLayoutKV = typename Kernel_traits::SmemLayoutKV;

    extern __shared__ char smem[];

    Tensor gQ = make_tensor(make_gmem_ptr(reinterpret_cast<T*>(params.q_ptr)),
                    make_shape(Int<M>{}, Int<K>{}),
                    make_stride(Int<K>{}, Int<1>{}));
    Tensor sQ = make_tensor(make_smem_ptr(reinterpret_cast<T*>(smem)),
                            SmemLayoutQ{});

    Tensor gK = make_tensor(make_gmem_ptr(reinterpret_cast<T*>(params.k_ptr)),
                    make_shape(Int<N>{}, Int<K>{}),
                    make_stride(Int<K>{}, Int<1>{}));
    
    Tensor sK = make_tensor(sQ.data() + size(sQ),
                            SmemLayoutKV{});
    
    const int idx = threadIdx.x;

    GmemTiledCopyQKV gmem_tiled_copy_QKV;
    auto gmem_thr_copy_QKV = gmem_tiled_copy_QKV.get_slice(idx);
    Tensor tQgQ = gmem_thr_copy_QKV.partition_S(gQ);
    Tensor tQsQ = gmem_thr_copy_QKV.partition_D(sQ);
    Tensor tKgK = gmem_thr_copy_QKV.partition_S(gK);  
    Tensor tKsK = gmem_thr_copy_QKV.partition_D(sK);

    typename Kernel_traits::TiledMma tiled_mma;
    auto thr_mma = tiled_mma.get_thread_slice(idx);
    Tensor tSrQ  = thr_mma.partition_fragment_A(sQ);                           // (MMA,MMA_M,MMA_K)
    Tensor tSrK  = thr_mma.partition_fragment_B(sK);                           // (MMA,MMA_N,MMA_K)

    auto smem_tiled_copy_Q = make_tiled_copy_A(typename Kernel_traits::SmemCopyAtom{}, tiled_mma);
    auto smem_thr_copy_Q = smem_tiled_copy_Q.get_thread_slice(idx);
    Tensor tSsQ = smem_thr_copy_Q.partition_S(sQ);
    auto smem_tiled_copy_K = make_tiled_copy_B(typename Kernel_traits::SmemCopyAtom{}, tiled_mma);
    auto smem_thr_copy_K = smem_tiled_copy_K.get_thread_slice(idx);
    Tensor tSsK = smem_thr_copy_K.partition_S(sK);

    Tensor acc_s = partition_fragment_C(tiled_mma, Shape<Int<M>, Int<N>>{});  // (MMA=4, MMA_M, MMA_N)
    Tensor acc_o = partition_fragment_C(tiled_mma, Shape<Int<M>, Int<K>>{});  // MMA, MMA_M, MMA_K
    clear(acc_s);
    clear(acc_o);
    flash::Softmax<2 * size<1>(acc_s)> softmax;
    
    
    // global to shared memory
    for (int m = 0; m < size<1>(tQgQ); m++) {
        for (int k = 0; k < size<2>(tQgQ); k++) {
            copy(gmem_tiled_copy_QKV, tQgQ(_, m, k), tQsQ(_, m, k));
        }
    }

    for (int m = 0; m < size<1>(tKgK); m++) {
        for (int k = 0; k < size<2>(tKgK); k++) {
            copy(gmem_tiled_copy_QKV, tKgK(_, m, k), tKsK(_, m, k));
        }
    }

    cp_async_fence();

    cp_async_wait<0>();

    __syncthreads();

    if (thread0()) {
        print("gQ: "); print(gQ); print("\n");
        print("sQ: "); print(sQ); print("\n");
        print("acc_s: "); print(acc_s); print("\n");
        print("tSsQ: "); print(tSsQ); print("\n");
        print_tensor(tSsQ);
        print("tSsK: "); print(tSsK); print("\n");
        print_tensor(tSsK);
    }
    

    CUTE_STATIC_ASSERT_V(size<1>(tSrQ) == size<1>(acc_s));                     // MMA_M
    CUTE_STATIC_ASSERT_V(size<1>(tSrK) == size<2>(acc_s));                     // MMA_N
    CUTE_STATIC_ASSERT_V(size<2>(tSrQ) == size<2>(tSrK));                     // MMA_K
    Tensor tCrA_copy_view = smem_thr_copy_Q.retile_D(tSrQ);
    CUTE_STATIC_ASSERT_V(size<1>(tSsQ) == size<1>(tCrA_copy_view));            // M
    Tensor tCrB_copy_view = smem_thr_copy_K.retile_D(tSrK);
    CUTE_STATIC_ASSERT_V(size<1>(tSsK) == size<1>(tCrB_copy_view));            // N
    cute::copy(smem_tiled_copy_Q, tSsQ(_, _, _0{}), tCrA_copy_view(_, _, _0{})); 
    cute::copy(smem_tiled_copy_K, tSsK(_, _, _0{}), tCrB_copy_view(_, _, _0{}));
    
    if (thread0()) {
        print("tSrQ: "); print(tSrQ); print("\n");
        print_tensor(tSrQ);
        print("tSrK: "); print(tSrK); print("\n");
        print_tensor(tSrK);
    }

    if (thread0()) {
        printf("tCrA_copy_view: "); print(tCrA_copy_view); printf("\n");
        print_tensor(tCrA_copy_view);
        printf("tCrB_copy_view: "); print(tCrB_copy_view); printf("\n");
        print_tensor(tCrB_copy_view);
    }

    #pragma unroll
    for (int i = 0; i < size<2>(tSrQ); ++i) {
        if (i < size<2>(tSrQ) - 1) {
            cute::copy(smem_tiled_copy_Q, tSsQ(_, _, i + 1), tCrA_copy_view(_, _, i + 1));
            cute::copy(smem_tiled_copy_K, tSsK(_, _, i + 1), tCrB_copy_view(_, _, i + 1)); 
        }
        cute::gemm(tiled_mma, tSrQ(_, _, i), tSrK(_, _, i), acc_s);
    }

    __syncthreads();

    if (thread0()) {
        printf("acc_s: "); print(acc_s); print("\n");
        print_tensor(acc_s);
        for (int i = 0; i < size(acc_s); i++) {
            print(acc_s.data()[i]); printf("\n");
        }
    }
    softmax.template softmax_rescale_o</*Is_first=*/true,  /*Check_inf=*/false>(acc_s, acc_o, params.scale_softmax_log2);
    Tensor lse = softmax.template normalize_softmax_lse(acc_s, params.scale_softmax);
    // Convert acc_s from fp32 to fp16/bf16
    constexpr int numel = decltype(size(acc_s))::value;
    cutlass::NumericArrayConverter<T, float, numel> convert_op;
    auto frag = convert_op(*reinterpret_cast<const cutlass::Array<float, numel> *>(acc_s.data()));
    Tensor rO = make_tensor(make_rmem_ptr<T>(&frag), acc_s.layout());

    if (thread0()) {
        printf("rO: "); print(rO); print("\n");
        print_tensor(rO);
    }
    Tensor sO = make_tensor(sQ.data(), typename Kernel_traits::SmemLayoutO{});    // (SMEM_M,SMEM_N)
    auto smem_tiled_copy_O = make_tiled_copy_C(typename Kernel_traits::SmemCopyAtomO{}, tiled_mma);
    auto smem_thr_copy_O = smem_tiled_copy_O.get_thread_slice(idx);
    Tensor taccOrO = smem_thr_copy_O.retile_S(rO);        // ((Atom,AtomNum), MMA_M, MMA_N)
    Tensor taccOsO = smem_thr_copy_O.partition_D(sO);     // ((Atom,AtomNum),PIPE_M,PIPE_N)
    if (thread0()) {
        printf("sO: "); print(sO); print("\n");
        printf("taccOrO: "); print(taccOrO); print("\n");
        printf("taccOsO: "); print(taccOsO); print("\n");
    }
    // copy to shared memory
    cute::copy(smem_tiled_copy_O, taccOrO, taccOsO);

    __syncthreads();

    if (thread0()) {
        printf("sO: \n");
        print_tensor(sO);
    }

    
    // shared memory to register
    Tensor gO = make_tensor(make_gmem_ptr(reinterpret_cast<T*>(params.o_ptr)),
                    make_shape(Int<M>{}, Int<N>{}),
                    make_stride(Int<N>{}, Int<1>{}));
    
    typename Kernel_traits::GmemTiledCopyO gmem_tiled_copy_O;
    auto gmem_thr_copy_O = gmem_tiled_copy_O.get_thread_slice(idx);
    Tensor tOsO = gmem_thr_copy_O.partition_S(sO);        // ((Atom,AtomNum),ATOM_M,ATOM_N)
    Tensor tOgO = gmem_thr_copy_O.partition_D(gO);
    
    //__syncthreads();
    
    Tensor tOrO = make_tensor<T>(shape(tOgO));
    cute::copy(gmem_tiled_copy_O, tOsO, tOrO);
    if (thread0()) {
        printf("tOsO: "); print(tOsO); printf("\n");
        print_tensor(tOsO);
        printf("tOrO: "); print(tOrO); printf("\n");
        print_tensor(tOrO);
        printf("tOgO: "); print(tOgO); printf("\n");
    }

    // register to global memory
    for (int m = 0; m < size<1>(tOrO); m++) {
        for (int k = 0; k < size<2>(tOrO); k++) {
            cute::copy(gmem_tiled_copy_O, tOrO(_, m, k), tOgO(_, m, k));
        }
    }

}



template <typename T>
void qk_matmul_kernel_launch(const at::Tensor& q, const at::Tensor& k, at::Tensor& o, float softmax_scale) {
    const int M = q.size(0);
    const int K = q.size(1);
    const int N = k.size(0);
    printf("m %d, n %d, k %d\n", M, N, K);
    TORCH_CHECK(K == 128, "only support k == 128");
    fwd_params params;
    params.q_ptr = q.data_ptr();
    params.k_ptr = k.data_ptr();
    params.o_ptr = o.data_ptr();
    params.scale_softmax = softmax_scale;
    params.scale_softmax_log2 = softmax_scale * M_LOG2E;
    auto stream = at::cuda::getCurrentCUDAStream().stream();
    Kernel_traits<T, 128, 64, 128> config;
    auto kernel = &qk_matmul_kernel<decltype(config)>;
    const int smem_size = config.kSmemSize;
    printf("smem_size is %d\n", smem_size);
    if (smem_size >= 48 * 1024) {
        hipFuncSetAttribute(reinterpret_cast<const void*>(kernel), hipFuncAttributeMaxDynamicSharedMemorySize, smem_size);
    }
    kernel<<<1, config.kNThreads, smem_size, stream>>>(params);
}


void qk_matmul(
    const at::Tensor& q,
    const at::Tensor& k,
    at::Tensor& o,
    const float softmax_scale
) {
    at::cuda::CUDAGuard device_guard{q.device()};
    auto q_dtype = q.dtype();
    TORCH_CHECK(q_dtype == torch::kFloat16 || q_dtype == torch::kBFloat16, "only support fp16 and bf16 data type");
    TORCH_CHECK(k.dtype() == q_dtype, "query and key must have the same dtype");
    TORCH_CHECK(q.stride(-1) == 1, "Input tensor must have contiguous last dimension");
    TORCH_CHECK(k.stride(-1) == 1, "Input tensor must have contiguous last dimension");
     
    FP16_SWITCH(q_dtype != torch::kBFloat16, [&] {
        qk_matmul_kernel_launch<elem_type>(q, k, o, softmax_scale);
    });

}